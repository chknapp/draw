#include "hip/hip_runtime.h"

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "../toefl/inc/dg/timer.cuh"
#include "device_window.cuh"
/**
 * @brief Functor returning a gaussian
 * \f[
   f(x,y) = Ae^{-(\frac{(x-x_0)^2}{2\sigma_x^2} + \frac{(y-y_0)^2}{2\sigma_y^2}} 
   \f]
 */
struct Gaussian
{
    /**
     * @brief Functor returning a gaussian
     *
     * @param x0 x-center-coordinate
     * @param y0 y-center-coordinate
     * @param sigma_x x - variance
     * @param sigma_y y - variance 
     * @param amp Amplitude
     */
    Gaussian( float x0, float y0, float sigma_x, float sigma_y, float amp)
        : x00(x0), y00(y0), sigma_x(sigma_x), sigma_y(sigma_y), amplitude(amp){}
    /**
     * @brief Return the value of the gaussian
     *
     * \f[
       f(x,y) = Ae^{-(\frac{(x-x_0)^2}{2\sigma_x^2} + \frac{(y-y_0)^2}{2\sigma_y^2}} 
       \f]
     * @param x x - coordinate
     * @param y y - coordinate
     *
     * @return gaussian
     */
    float operator()(float x, float y)
    {
        return  amplitude*
                   exp( -((x-x00)*(x-x00)/2./sigma_x/sigma_x +
                          (y-y00)*(y-y00)/2./sigma_y/sigma_y) );
    }
  private:
    float  x00, y00, sigma_x, sigma_y, amplitude;

};

const unsigned Nx = 7000, Ny = 4000;
const float lx = 2., ly = 1.;
const float hx = lx/(float)Nx, hy = ly/(float)Ny;

int main()
{
    //Create Window and set window title
    draw::DeviceWindow w( 800, 400);
    // generate a vector on the grid to visualize 
    Gaussian g( 1.2, 0.3, .1, .1, 1);
    thrust::host_vector<float> visual(Nx*Ny);
    for(unsigned i=0; i<Ny; i++)
        for( unsigned j=0; j<Nx; j++)
            visual[i*Nx+j] = -g( (float)j*hx, (float)i*hy);
    thrust::device_vector<float> dvisual = visual;

    //create a colormap
    draw::ColorMapRedBlueExt colors( 1.);
    //set scale
    colors.scale() =  1.;

    int running = GL_TRUE;
    dg::Timer t;
    while (running)
    {
        w.title() << "Hello world\n";
        t.tic();
        w.draw( dvisual, Nx, Ny, colors);
        t.toc();
        std::cout << "Drawing took "<<t.diff()*1000.<<"ms\n";
        glfwWaitEvents();
        running = !glfwGetKey( GLFW_KEY_ESC) &&
                    glfwGetWindowParam( GLFW_OPENED);
    }

    return 0;
}
